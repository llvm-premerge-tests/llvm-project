// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=4 -DUSER -x hip -o %t_4 %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=5 -DUSER -x hip -o %t_5 %s

// RUN: %clang_cc1 -fcuda-is-device -triple amdgcn-amd-amdhsa -emit-llvm \
// RUN:   -mcode-object-version=none -DDEVICELIB -x hip -o %t_0 %s

// RUN: llvm-link %t_0 %t_4 -o -| llvm-dis -o - | FileCheck -check-prefix=LINKED4 %s
// RUN: llvm-link %t_0 %t_5 -o -| llvm-dis -o - | FileCheck -check-prefix=LINKED5 %s

#include "Inputs/hip/hip_runtime.h"

// LINKED4: llvm.amdgcn.abi.version = weak_odr hidden addrspace(4) constant i32 400, align 4
// LINKED4-LABEL: bar
// LINKED4: load i32, ptr addrspacecast (ptr addrspace(4) @llvm.amdgcn.abi.version to ptr), align {{.*}}
// LINKED4: [[ABI5_X:%.*]] = icmp sge i32 %{{.*}}, 500
// LINKED4: call align 8 dereferenceable(256) ptr addrspace(4) @llvm.amdgcn.implicitarg.ptr()
// LINKED4: [[GEP_5_X:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 12
// LINKED4: call align 4 dereferenceable(64) ptr addrspace(4) @llvm.amdgcn.dispatch.ptr()
// LINKED4: [[GEP_4_X:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 4
// LINKED4: select i1 [[ABI5_X]], ptr addrspace(4) [[GEP_5_X]], ptr addrspace(4) [[GEP_4_X]]
// LINKED4: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef
// LINKED4: "amdgpu_code_object_version", i32 400

// LINKED5: llvm.amdgcn.abi.version = weak_odr hidden addrspace(4) constant i32 500, align 4
// LINKED5-LABEL: bar
// LINKED5: load i32, ptr addrspacecast (ptr addrspace(4) @llvm.amdgcn.abi.version to ptr), align {{.*}}
// LINKED5: [[ABI5_X:%.*]] = icmp sge i32 %{{.*}}, 500
// LINKED5: call align 8 dereferenceable(256) ptr addrspace(4) @llvm.amdgcn.implicitarg.ptr()
// LINKED5: [[GEP_5_X:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 12
// LINKED5: call align 4 dereferenceable(64) ptr addrspace(4) @llvm.amdgcn.dispatch.ptr()
// LINKED5: [[GEP_4_X:%.*]] = getelementptr i8, ptr addrspace(4) %{{.*}}, i32 4
// LINKED5: select i1 [[ABI5_X]], ptr addrspace(4) [[GEP_5_X]], ptr addrspace(4) [[GEP_4_X]]
// LINKED5: load i16, ptr addrspace(4) %{{.*}}, align 2, !range [[$WS_RANGE:![0-9]*]], !invariant.load{{.*}}, !noundef
// LINKED5: "amdgpu_code_object_version", i32 500

#ifdef DEVICELIB
__device__ void bar(int *out)
{
  *out = __builtin_amdgcn_workgroup_size_x();
}
#endif

#ifdef USER
__device__ void bar(int *out);
__device__ void foo()
{
  int *out;
  bar(out);
}
#endif
